#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "cFunctions.h"
#define MAX_THREADS 1024

                        
__global__ void calcScore(Score* deviceAllScores, char* deviceSeq1, char* deviceSeq2, int lenghtOfSeq2, int threadsInBlock, int allScoresOffset, int* deviceScoreMat)
{     
    int i,j;
    int count = 0; 
    int loc = 0;
    int scoreIndex = blockIdx.y * gridDim.x + blockIdx.x + allScoresOffset; 
    extern __shared__ int shared_arr[];
    
    while (loc + threadIdx.x < lenghtOfSeq2)
    {     	            		   
        j = deviceSeq2[threadIdx.x+loc] - 'A';
        if (threadIdx.x+loc >= blockIdx.y + 1) 
            i = deviceSeq1[threadIdx.x+loc+blockIdx.x+1] - 'A';
        else
            i = deviceSeq1[threadIdx.x+loc+blockIdx.x] - 'A';
        shared_arr[threadIdx.x+loc] = deviceScoreMat[i*ABC_NUMBER + j];
        count++;
        loc = count*threadsInBlock;          
    }  
    
    __syncthreads();

    if(threadIdx.x == 0) 
    {
        deviceAllScores[scoreIndex].k = blockIdx.y + 1; 
        deviceAllScores[scoreIndex].n = blockIdx.x; 
        for (int i = 0; i < lenghtOfSeq2; i++)
            deviceAllScores[scoreIndex].scoreWeight+= shared_arr[i];              
    }
    
}

int checkStatus(hipError_t cudaStatus, int* deviceScoreMat, char* deviceSeq1, char* deviceSeq2, score* deviceAllScores, std::string err)
{
    if(cudaStatus != hipSuccess)
    {
        std::cout << err <<std::endl;

        free(deviceScoreMat);
        free(deviceSeq1);        
        free(deviceSeq2);    
        free(deviceAllScores);
            
        return 0;
    }
    return 1; 
}


int initCudaCalcs(char** seq2Arr, int seq2ArrSize, score* allScoresFromCuda, int* allScoresFromCudaBySize, char* seq1, int** scoreMat)
{
    hipError_t cudaStatus;
    int* deviceScoreMat = 0;
    char* deviceSeq1 = 0;  
    char* deviceSeq2 = 0; 
    int lenghtOfSeq1, lenghtOfSeq2;
    Score* deviceAllScores = 0;
    int allScoresOffset = 0;
    int threadsInBlock = 0;
    int* scoreArr = (int*)malloc(sizeof(int)* ABC_NUMBER * ABC_NUMBER); 
    scoreMatToArray(scoreMat, scoreArr);
    
    cudaStatus = hipMalloc((void**)&deviceScoreMat, ABC_NUMBER * ABC_NUMBER * sizeof(int));
    if(!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipMalloc have failed!"))
        return 0;
    cudaStatus = hipMemcpy(deviceScoreMat, scoreArr, ABC_NUMBER * ABC_NUMBER * sizeof(int), hipMemcpyHostToDevice);
    if(!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipMemcpy have failed!"))
        return 0; 
 
    int allSize = 0;
    for (int i = 0; i < seq2ArrSize; i++)
        allSize += allScoresFromCudaBySize[i];
    cudaStatus = hipMalloc((void**)&deviceAllScores, sizeof(score) * allSize);
    if(!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipMalloc have failed!"))
        return 0;
    cudaStatus = hipMemcpy(deviceAllScores, allScoresFromCuda, sizeof(score) * allSize, hipMemcpyHostToDevice);
    if(!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipMemcpy have failed!"))
        return 0;
     
    lenghtOfSeq1 = strlen(seq1);
    cudaStatus = hipMalloc((void**)&deviceSeq1, sizeof(char)* (lenghtOfSeq1 + 1));
    if(!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipMalloc have failed!"))
        return 0;
    cudaStatus = hipMemcpy(deviceSeq1, seq1, sizeof(char)* (lenghtOfSeq1 + 1), hipMemcpyHostToDevice);
    if(!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipMemcpy have failed!"))
        return 0;
      
    for (int i = 0; i < seq2ArrSize; i++)
    {
        lenghtOfSeq2 = strlen(seq2Arr[i]); 
        cudaStatus = hipMalloc((void**)&deviceSeq2, sizeof(char)* (lenghtOfSeq2 + 1));
        if(!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipMalloc have failed!"))
            return 0;  
        cudaStatus = hipMemcpy(deviceSeq2, seq2Arr[i], sizeof(char)* (lenghtOfSeq2 + 1), hipMemcpyHostToDevice);
        if(!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipMemcpy have failed!"))
            return 0;
         
        dim3 numBlocks (lenghtOfSeq1 - lenghtOfSeq2, lenghtOfSeq2);
        if (MAX_THREADS < lenghtOfSeq2)
            threadsInBlock = MAX_THREADS;
        else
            threadsInBlock = lenghtOfSeq2;
    
        calcScore<<<numBlocks,threadsInBlock,lenghtOfSeq2*sizeof(int)>>>(deviceAllScores, deviceSeq1, deviceSeq2, lenghtOfSeq2, threadsInBlock, allScoresOffset, deviceScoreMat);
        cudaStatus = hipDeviceSynchronize();
        if(!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "calculateAlignmentScores have failed!"))
            return 0;
        
        allScoresOffset += allScoresFromCudaBySize[i];
    }
    
    cudaStatus = hipMemcpy(allScoresFromCuda, deviceAllScores, sizeof(score) * allSize, hipMemcpyDeviceToHost);
    if(!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipMemcpy have failed!"))
        return 0;
  
    cudaStatus = hipFree(deviceSeq1);
    if (!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipFree have failed!"))
        return 0;    
    cudaStatus = hipFree(deviceSeq2);
    if (!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipFree have failed!"))
        return 0;   
    cudaStatus = hipFree(deviceAllScores);
    if (!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipFree have failed!"))
        return 0;
     cudaStatus = hipFree(deviceScoreMat);
    if (!checkStatus(cudaStatus, deviceScoreMat, deviceSeq1, deviceSeq2, deviceAllScores, "hipFree have failed!"))
       return 0;    
 
    return 1; 
}


